
#include <hip/hip_runtime.h>
/*! \page Numerics SPH Dam break simulation with Dynamic load balacing on Multi-GPU
 *
 *
 * [TOC]
 *
 *
 * # SPH with Dynamic load Balancing on GPU # {#SPH_dlb_gpu}
 *
 *
 * This example show the classical SPH Dam break simulation with load balancing and dynamic load balancing. The main difference with
 * \ref SPH_dlb is that here we use GPUs and 1.2 Millions particles.
 *
 * \htmlonly
 * <a href="#" onclick="hide_show('vector-video-3')" >Simulation video 1</a><br>
 * <div style="display:none" id="vector-video-3">
 * <video id="vid3" width="1200" height="576" controls> <source src="http://openfpm.mpi-cbg.de/web/images/examples/7_SPH_dlb/sph_gpu1.mp4" type="video/mp4"></video>
 * </div>
 * <a href="#" onclick="hide_show('vector-video-4')" >Simulation video 2</a><br>
 * <div style="display:none" id="vector-video-4">
 * <video id="vid4" width="1200" height="576" controls> <source src="http://openfpm.mpi-cbg.de/web/images/examples/7_SPH_dlb/sph_gpu2.mp4" type="video/mp4"></video>
 * </div>
 * <a href="#" onclick="hide_show('vector-video-15')" >Simulation video 3</a><br>
 * <div style="display:none" id="vector-video-15">
 * <video id="vid15" width="1200" height="576" controls> <source src="http://openfpm.mpi-cbg.de/web/images/examples/7_SPH_dlb/sph_gpu3.mp4" type="video/mp4"></video>
 * </div>
 * \endhtmlonly
 *
 * This example use all the features explained in example \ref e3_md_gpu. Additionally this example show how to remove particles
 * on GPU using a bulk remove function on GPU
 *
 * ## Bulk remove
 *
 * On SPH we have the necessity to remove particles that go out of bound. OpenFPM provide the function \b remove_marked \b .
 *
 * \snippet example/Numerics/Odeint/SPH_dlb_gpu/main.cu remove_marked_part
 *
 * where vectorDist is the vector_dist_gpu red is the property that mark which particle must be removed. We mark the particle to be removed in the function kernel
 * We check if the particle go out of the region of interest or their density go critically far from the rest density
 *
 * \snippet example/Numerics/Odeint/SPH_dlb_gpu/main.cu mark_to_remove_kernel
 *
 * ## Macro CUDA_LAUNCH
 *
 * When we want to launch a kernel "my_kernel" on CUDA we in general use the Nvidia CUDA syntax
 *
 * my_kernel<<<wthr,thr>>>(arguments ... )
 *
 * Where wthr is the number of workgroups and thr is the number of threads in a workgroup and arguments... are the arguments to pass to the kernel.
 * Equivalently we can launch a kernel with the macro CUDA_LAUNCH_DIM3(my_kernel,wthr,thr,arguments...) or CUDA_LAUNCH(my_kernel,ite,arguments) where
 * ite has been taken using getDomainIteratorGPU. There are several advantage on using CUDA_LAUNCH. The first advantage in using the macro is enabling SE_CLASS1
 * all kernel launch become synchronous and an error check is performed before continue to the next kernel making debugging easier. Another feature is the possibility
 * to run CUDA code on CPU without a GPU. compiling with "CUDA_ON_CPU=1 make" (Note openfpm must be compiled with GPU support (-g)  or with CUDA_ON_CPU support
 * (-c "... --enable_cuda_on_cpu"). You can compile this example on CPU. You do not have to change a single line of code for this example. (Check the video to see this
 * feature in action). All the openfpm GPU example and CUDA example can run on CPU if they use CUDA_LAUNCH as macro. We are planning to support
 * AMD GPUs as well using this system.
 *
 * \include example/Numerics/Odeint/SPH_dlb_gpu/main.cu
 *
 */

#ifdef __NVCC__

#include <math.h>

#include "Vector/vector_dist.hpp"
#include "Draw/DrawParticles.hpp"
#include "OdeIntegrators/OdeIntegrators.hpp"
#include "Operators/Vector/vector_dist_operators.hpp"

typedef float real_number;

// A constant to indicate boundary particles
#define BOUNDARY 0

// A constant to indicate fluid particles
#define FLUID 1

// initial spacing between particles dp in the formulas
const real_number dp = 0.0085;
// Maximum height of the fluid water
// is going to be calculated and filled later on
real_number h_swl = 0.0;

// c_s in the formulas (constant used to calculate the sound speed)
const real_number coeff_sound = 20.0;

// gamma in the formulas
const real_number gamma_ = 7.0;

// sqrt(3.0*dp*dp) support of the kernel
const real_number H = 0.0147224318643;

// Eta in the formulas
const real_number Eta2 = 0.01 * H*H;

// alpha in the formula
const real_number visco = 0.1;

// cbar in the formula (calculated later)
real_number cbar = 0.0;

// Mass of the fluid particles
const real_number MassFluid = 0.000614125;

// Mass of the boundary particles
const real_number MassBound = 0.000614125;

// End simulation time
#ifdef TEST_RUN
const real_number t_end = 0.001;
#else
const real_number t_end = 1.5;
#endif

// Gravity acceleration
const real_number gravity = 9.81;

// Reference densitu 1000Kg/m^3
const real_number RhoZero = 1000.0;

// Filled later require h_swl, it is b in the formulas
real_number B = 0.0;

// Constant used to define time integration
const real_number CFLnumber = 0.2;

// Minimum T
const real_number DtMin = 0.00001;

// Minimum Rho allowed
const real_number RhoMin = 700.0;

// Maximum Rho allowed
const real_number RhoMax = 1300.0;

// Filled in initialization
real_number max_fluid_height = 0.0;

// Properties

// FLUID or BOUNDARY
const size_t TYPE = 0;

// Density
const int RHO = 1;

// Density at step n-1
const int RHO_PREV = 2;

// Pressure
const int PRESSURE = 3;

// Delta rho calculated in the force calculation
const int DRHO = 4;

// calculated force
const int FORCE = 5;

// velocity
const int VELOCITY = 6;

// velocity at previous step
const int VELOCITY_PREV = 7;

// temporal variable to store velocity
const int VELOCITY_TMP = 11;

// temporal variable to store density
const int RHO_TMP = 10;

const int RED = 8;

const int RED2 = 9;

// Type of the vector containing particles
typedef vector_dist_gpu<3,real_number,aggregate<size_t,real_number,  real_number,    real_number,     real_number,     VectorS<3, real_number>, VectorS<3, real_number>, VectorS<3, real_number>, real_number, real_number, real_number, VectorS<3, real_number>>> particles;
//                                              |          |             |               |                |                      |                         |                        |                  |           |			|				|
//                                              |          |             |               |                |                      |                         |                        |                  |           |			|				|
//                                             type      density       density        Pressure          delta                  force                    velocity                 velocity           reduction    another	 temp density temp velocity
//                                                                     at n-1                           density                                                                  at n - 1           buffer   reduction buffer


struct ModelCustom
{
	template<typename Decomposition, typename vector>
	inline void addComputation(
		Decomposition & dec,
		vector & vectorDist,
		size_t v,
		size_t p)
	{
		if (vectorDist.template getProp<TYPE>(p) == FLUID)
			dec.addComputationCost(v,4);
		else
			dec.addComputationCost(v,3);
	}

	template<typename Decomposition> inline void applyModel(Decomposition & dec, size_t v)
	{
		dec.setSubSubDomainComputationCost(v, dec.getSubSubDomainComputationCost(v) * dec.getSubSubDomainComputationCost(v));
	}

	real_number distributionTol()
	{
		return 1.01;
	}
};

template<typename vd_type>
__global__ void EqState_gpu(vd_type vectorDist, real_number B)
{
	auto a = GET_PARTICLE(vectorDist);

	real_number rho_a = vectorDist.template getProp<RHO>(a);
	real_number rho_frac = rho_a / RhoZero;

	vectorDist.template getProp<PRESSURE>(a) = B*( rho_frac*rho_frac*rho_frac*rho_frac*rho_frac*rho_frac*rho_frac - 1.0);
}

inline void EqState(particles & vectorDist)
{
	auto it = vectorDist.getDomainIteratorGPU();

	CUDA_LAUNCH(EqState_gpu,it,vectorDist.toKernel(),B);
}


const real_number a2 = 1.0/M_PI/H/H/H;

inline __device__ __host__ real_number Wab(real_number r)
{
	r /= H;

	if (r < 1.0)
		return (1.0 - 3.0/2.0*r*r + 3.0/4.0*r*r*r)*a2;
	else if (r < 2.0)
		return (1.0/4.0*(2.0 - r)*(2.0 - r)*(2.0 - r))*a2;
	else
		return 0.0;
}


const real_number c1 = -3.0/M_PI/H/H/H/H;
const real_number d1 = 9.0/4.0/M_PI/H/H/H/H;
const real_number c2 = -3.0/4.0/M_PI/H/H/H/H;
const real_number a2_4 = 0.25*a2;
// Filled later
real_number W_dap = 0.0;

inline __device__ __host__ void DWab(Point<3,real_number> & dx, Point<3,real_number> & DW, real_number r)
{
	const real_number qq=r/H;

    real_number qq2 = qq * qq;
    real_number fac1 = (c1*qq + d1*qq2)/r;
    real_number b1 = (qq < 1.0f)?1.0f:0.0f;

    real_number wqq = (2.0f - qq);
    real_number fac2 = c2 * wqq * wqq / r;
    real_number b2 = (qq >= 1.0f && qq < 2.0f)?1.0f:0.0f;

    real_number factor = (b1*fac1 + b2*fac2);

    DW.get(0) = factor * dx.get(0);
    DW.get(1) = factor * dx.get(1);
    DW.get(2) = factor * dx.get(2);
}

// Tensile correction
inline __device__ __host__  real_number Tensile(real_number r, real_number rhoa, real_number rhob, real_number prs1, real_number prs2, real_number W_dap)
{
	const real_number qq=r/H;
	//-Cubic Spline kernel
	real_number wab;
	if(r>H)
	{
		real_number wqq1=2.0f-qq;
		real_number wqq2=wqq1*wqq1;

		wab=a2_4*(wqq2*wqq1);
	}
	else
	{
	    real_number wqq2=qq*qq;
	    real_number wqq3=wqq2*qq;

	    wab=a2*(1.0f-1.5f*wqq2+0.75f*wqq3);
	}

	//-Tensile correction.
	real_number fab=wab*W_dap;
	fab*=fab; fab*=fab; //fab=fab^4
	const real_number tensilp1=(prs1/(rhoa*rhoa))*(prs1>0.0f? 0.01f: -0.2f);
	const real_number tensilp2=(prs2/(rhob*rhob))*(prs2>0.0f? 0.01f: -0.2f);

	return (fab*(tensilp1+tensilp2));
}


inline __device__ __host__ real_number Pi(const Point<3,real_number> & dr, real_number rr2, Point<3,real_number> & dv, real_number rhoa, real_number rhob, real_number massb, real_number cbar, real_number & visc)
{
	const real_number dot = dr.get(0)*dv.get(0) + dr.get(1)*dv.get(1) + dr.get(2)*dv.get(2);
	const real_number dot_rr2 = dot/(rr2+Eta2);
	visc=(dot_rr2 < visc)?visc:dot_rr2;

	if(dot < 0)
	{
		const float amubar=H*dot_rr2;
		const float robar=(rhoa+rhob)*0.5f;
		const float pi_visc=(-visco*cbar*amubar/robar);

		return pi_visc;
    }
	else
		return 0.0f;
}

template<typename particles_type, typename CellList_type>
__global__ void calc_forces_gpu(particles_type vectorDist, CellList_type cellList, real_number W_dap, real_number cbar)
{
	auto a = GET_PARTICLE(vectorDist);

	real_number max_visc = 0.0f;

	// Get the position xp of the particle
	Point<3,real_number> xa = vectorDist.getPos(a);

	// Type of the particle
	unsigned int typea = vectorDist.template getProp<TYPE>(a);

	// Get the density of the of the particle a
	real_number rhoa = vectorDist.template getProp<RHO>(a);

	// Get the pressure of the particle a
	real_number Pa = vectorDist.template getProp<PRESSURE>(a);

	// Get the Velocity of the particle a
	Point<3,real_number> va = vectorDist.template getProp<VELOCITY>(a);

	// Reset the force counter (- gravity on zeta direction)
	Point<3,real_number> force_;
	force_.get(0) = 0.0f;
	force_.get(1) = 0.0f;
	force_.get(2) = -gravity;
	real_number drho_ = 0.0f;

	// Get an iterator over the neighborhood particles of p
	auto Np = cellList.getNNIteratorBox(cellList.getCell(xa));

	// For each neighborhood particle
	while (Np.isNext() == true)
	{
		// ... q
		auto b = Np.get();

		// Get the position xp of the particle
		Point<3,real_number> xb = vectorDist.getPos(b);

		if (a == b)	{++Np; continue;};

		unsigned int typeb = vectorDist.template getProp<TYPE>(b);

		real_number massb = (typeb == FLUID)?MassFluid:MassBound;
		Point<3,real_number> vb = vectorDist.template getProp<VELOCITY>(b);
		real_number Pb = vectorDist.template getProp<PRESSURE>(b);
		real_number rhob = vectorDist.template getProp<RHO>(b);

		// Get the distance between p and q
		Point<3,real_number> dr = xa - xb;
		// take the norm of this vector
		real_number r2 = norm2(dr);

		// if they interact
		if (r2 < 4.0*H*H && r2 >= 1e-16)
		{
			real_number r = sqrt(r2);

			Point<3,real_number> v_rel = va - vb;

			Point<3,real_number> DW;
			DWab(dr,DW,r);

			real_number factor = - massb*((Pa + Pb) / (rhoa * rhob) + Tensile(r,rhoa,rhob,Pa,Pb,W_dap) + Pi(dr,r2,v_rel,rhoa,rhob,massb,cbar,max_visc));

			// Bound - Bound does not produce any change
			// factor = (typea == BOUNDARY && typeb == BOUNDARY)?0.0f:factor;
			factor = (typea != FLUID)?0.0f:factor;

			force_.get(0) += factor * DW.get(0);
			force_.get(1) += factor * DW.get(1);
			force_.get(2) += factor * DW.get(2);

			real_number scal = massb*(v_rel.get(0)*DW.get(0)+v_rel.get(1)*DW.get(1)+v_rel.get(2)*DW.get(2));
			scal = (typea == BOUNDARY && typeb == BOUNDARY)?0.0f:scal;

			drho_ += scal;
		}

		++Np;
	}

	vectorDist.template getProp<RED>(a) = max_visc;

	vectorDist.template getProp<FORCE>(a)[0] = force_.get(0);
	vectorDist.template getProp<FORCE>(a)[1] = force_.get(1);
	vectorDist.template getProp<FORCE>(a)[2] = force_.get(2);
	vectorDist.template getProp<DRHO>(a) = drho_;
}

template<typename CellList> inline void calc_forces(particles & vectorDist, CellList & cellList, real_number & max_visc, size_t cnt)
{
	auto part = vectorDist.getDomainIteratorGPU(32);

	// Update the cell-list
	vectorDist.updateCellListGPU(cellList);

	CUDA_LAUNCH(calc_forces_gpu,part,vectorDist.toKernel(),cellList.toKernel(),W_dap,cbar);

	max_visc = reduce_local<RED,_max_>(vectorDist);
}

template<typename vector_type>
__global__ void max_acceleration_and_velocity_gpu(vector_type vectorDist)
{
	auto a = GET_PARTICLE(vectorDist);

	Point<3,real_number> acc(vectorDist.template getProp<FORCE>(a));
	vectorDist.template getProp<RED>(a) = norm(acc);

	Point<3,real_number> vel(vectorDist.template getProp<VELOCITY>(a));
	vectorDist.template getProp<RED2>(a) = norm(vel);
}

template<typename vector_type>
__global__ void checkGPU(vector_type vector)
{
	int i = threadIdx.x;
	printf("Check GPU %d %p %f\n", i, &vector.get<0>(i), vector.get<0>(i));
}



void max_acceleration_and_velocity(particles & vectorDist, real_number & max_acc, real_number & max_vel)
{
	// Calculate the maximum acceleration
	auto part = vectorDist.getDomainIteratorGPU();

	CUDA_LAUNCH(max_acceleration_and_velocity_gpu,part,vectorDist.toKernel());

	max_acc = reduce_local<RED,_max_>(vectorDist);
	max_vel = reduce_local<RED2,_max_>(vectorDist);

	Vcluster<> & v_cl = create_vcluster();
	v_cl.max(max_acc);
	v_cl.max(max_vel);
	v_cl.execute();
}


real_number calc_deltaT(particles & vectorDist, real_number ViscDtMax)
{
	real_number Maxacc = 0.0;
	real_number Maxvel = 0.0;
	max_acceleration_and_velocity(vectorDist,Maxacc,Maxvel);

	//-dt1 depends on force per unit mass.
	const real_number dt_f = (Maxacc)?sqrt(H/Maxacc):std::numeric_limits<float>::max();

	//-dt2 combines the Courant and the viscous time-step controls.
	const real_number dt_cv = H/(std::max(cbar,Maxvel*10.f) + H*ViscDtMax);

	//-dt new value of time step.
	real_number dt=real_number(CFLnumber)*std::min(dt_f,dt_cv);
	if(dt<real_number(DtMin))
	{dt=real_number(DtMin);}

	return dt;
}

template<typename vector_dist_type>
__global__ void checkPosPrpLimits(vector_dist_type vectorDist)
{
	auto p = GET_PARTICLE(vectorDist);

	// if the particle type is boundary
	if (vectorDist.template getProp<TYPE>(p) == BOUNDARY)
	{
		real_number rho = vectorDist.template getProp<RHO>(p);
		if (rho < RhoZero)
			vectorDist.template getProp<RHO>(p) = RhoZero;

		return;
	}

	// Check if the particle go out of range in space and in density, if they do mark them to remove it later
	if (vectorDist.getPos(p)[0] <  0.0 || vectorDist.getPos(p)[1] < 0.0 || vectorDist.getPos(p)[2] < 0.0 ||
		vectorDist.getPos(p)[0] >  1.61 || vectorDist.getPos(p)[1] > 0.68 || vectorDist.getPos(p)[2] > 0.50 ||
		vectorDist.template getProp<RHO>(p) < RhoMin || vectorDist.template getProp<RHO>(p) > RhoMax)
		{vectorDist.template getProp<RED>(p) = 1;}
}

size_t cnt = 0;

void verlet_int(particles & vectorDist, real_number dt)
{
	// particle iterator
	auto part = vectorDist.getDomainIteratorGPU();

	real_number dt205 = dt*dt*0.5;
	real_number dt2 = dt*2.0;

	auto posExpression = getV<POS_PROP, comp_dev>(vectorDist);
	auto posExpression2 = getV<POS_PROP>(vectorDist);
	auto forceExpression = getV<FORCE, comp_dev>(vectorDist);
	auto drhoExpression = getV<DRHO, comp_dev>(vectorDist);
	auto typeExpression = getV<TYPE, comp_dev>(vectorDist);
	auto velocityExpression = getV<VELOCITY, comp_dev>(vectorDist);

	auto rho_tmpExpression = getV<RHO_TMP, comp_dev>(vectorDist);
	auto rhoExpression = getV<RHO, comp_dev>(vectorDist);
	auto rho_prevExpression = getV<RHO_PREV, comp_dev>(vectorDist);

	auto velocity_prevExpression = getV<VELOCITY_PREV, comp_dev>(vectorDist);
	auto velocity_tmpExpression = getV<VELOCITY_TMP, comp_dev>(vectorDist);
	auto redExpression = getV<RED, comp_dev>(vectorDist);

	rho_tmpExpression = rhoExpression;
	rhoExpression = rho_prevExpression + dt2*drhoExpression;
	rho_prevExpression = rho_tmpExpression;

	posExpression = posExpression + velocityExpression*dt + forceExpression*dt205 * typeExpression;

	velocity_tmpExpression = velocityExpression;
	velocityExpression = velocity_prevExpression + forceExpression*dt2 * typeExpression;
	velocity_prevExpression = velocity_tmpExpression;

	redExpression = 0;

	CUDA_LAUNCH(checkPosPrpLimits,part,vectorDist.toKernel());

	// remove the particles marked
	remove_marked<RED>(vectorDist);

	// increment the iteration counter
	cnt++;
}


void euler_int(particles & vectorDist, real_number dt)
{

	// particle iterator
	auto part = vectorDist.getDomainIteratorGPU();

	real_number dt205 = dt*dt*0.5;

	auto posExpression = getV<POS_PROP, comp_dev>(vectorDist);
	auto forceExpression = getV<FORCE, comp_dev>(vectorDist);
	auto drhoExpression = getV<DRHO, comp_dev>(vectorDist);
	auto typeExpression = getV<TYPE, comp_dev>(vectorDist);

	auto rhoExpression = getV<RHO, comp_dev>(vectorDist);
	auto rho_prevExpression = getV<RHO_PREV, comp_dev>(vectorDist);

	auto velocityExpression = getV<VELOCITY, comp_dev>(vectorDist);
	auto velocity_prevExpression = getV<VELOCITY_PREV, comp_dev>(vectorDist);

	auto redExpression = getV<RED, comp_dev>(vectorDist);

	rho_prevExpression = rhoExpression;
	rhoExpression = rhoExpression + dt*drhoExpression;

	posExpression = posExpression + velocityExpression*dt + forceExpression*dt205 * typeExpression;

	velocity_prevExpression = velocityExpression;
	velocityExpression = velocityExpression + forceExpression*dt * typeExpression;

	redExpression = 0;

	CUDA_LAUNCH(checkPosPrpLimits,part,vectorDist.toKernel());

	// remove the particles
	remove_marked<RED>(vectorDist);

	cnt++;
}

template<typename vector_type, typename CellList_type>
__global__ void sensor_pressure_gpu(vector_type vectorDist, CellList_type cellList, Point<3,real_number> probe, real_number * press_tmp)
{
	real_number tot_ker = 0.0;

	// Get the position of the probe i
	Point<3,real_number> xp = probe;

	// get the iterator over the neighbohood particles of the probes position
	auto itg = cellList.getNNIteratorBox(cellList.getCell(xp));
	while (itg.isNext())
	{
		auto q = itg.get();

		// Only the fluid particles are importants
		if (vectorDist.template getProp<TYPE>(q) != FLUID)
		{
			++itg;
			continue;
		}

		// Get the position of the neighborhood particle q
		Point<3,real_number> xq = vectorDist.getPos(q);

		// Calculate the contribution of the particle to the pressure
		// of the probe
		real_number r = sqrt(norm2(xp - xq));

		real_number ker = Wab(r) * (MassFluid / RhoZero);

		// Also keep track of the calculation of the summed
		// kernel
		tot_ker += ker;

		// Add the total pressure contribution
		*press_tmp += vectorDist.template getProp<PRESSURE>(q) * ker;

		// next neighborhood particle
		++itg;
	}

	// We calculate the pressure normalizing the
	// sum over all kernels
	if (tot_ker == 0.0)
		{*press_tmp = 0.0;}
	else
		{*press_tmp = 1.0 / tot_ker * *press_tmp;}
}

template<typename Vector, typename CellList>
inline void sensor_pressure(Vector & vectorDist,
	CellList & cellList,
	openfpm::vector<openfpm::vector<real_number>> & press_t,
	openfpm::vector<Point<3,real_number>> & probes)
{
    Vcluster<> & v_cl = create_vcluster();

    press_t.add();

    for (size_t i = 0 ; i < probes.size() ; i++)
    {
    	// A float variable to calculate the pressure of the problem
    	CudaMemory press_tmp_(sizeof(real_number));
    	real_number press_tmp;

        // if the probe is inside the processor domain
		if (vectorDist.getDecomposition().isLocal(probes.get(i)) == true)
		{
			vectorDist.updateCellListGPU(cellList);

			Point<3,real_number> probe = probes.get(i);
			CUDA_LAUNCH_DIM3(sensor_pressure_gpu,1,1,vectorDist.toKernel(),cellList.toKernel(),probe,(real_number *)press_tmp_.toKernel());

			// move calculated pressure on
			press_tmp_.deviceToHost();
			press_tmp = *(real_number *)press_tmp_.getPointer();
		}

		// This is not necessary in principle, but if you
		// want to make all processor aware of the history of the calculated
		// pressure we have to execute this
		v_cl.sum(press_tmp);
		v_cl.execute();

		// We add the calculated pressure into the history
		press_t.last().add(press_tmp);
	}
}

int main(int argc, char* argv[])
{
    // initialize the library
	openfpm_init(&argc,&argv);

	// It contain for each time-step the value detected by the probes
	openfpm::vector<openfpm::vector<real_number>> press_t;
	openfpm::vector<Point<3,real_number>> probes;

	probes.add({0.8779f,0.3f,0.02f});
	probes.add({0.754f,0.31f,0.02f});

	// Here we define our domain a 2D box with internals from 0 to 1.0 for x and y
	Box<3,real_number> domain({-0.05f,-0.05f,-0.05f},{1.7010f,0.7065f,0.511f});
	size_t sz[3] = {207,90,66};

	// Fill W_dap
	W_dap = 1.0/Wab(H/1.5);

	// Here we define the boundary conditions of our problem
    size_t bc[3]={NON_PERIODIC,NON_PERIODIC,NON_PERIODIC};

	// extended boundary around the domain, and the processor domain
	Ghost<3,real_number> g(2*H);

	particles vectorDist(0,domain,bc,g,DEC_GRAN(512));

	//! \cond [draw fluid] \endcond

	// You can ignore all these dp/2.0 is a trick to reach the same initialization
	// of Dual-SPH that use a different criteria to draw particles
	Box<3,real_number> fluid_box({dp/2.0f,dp/2.0f,dp/2.0f},{0.4f+dp/2.0f,0.67f-dp/2.0f,0.3f+dp/2.0f});

	// return an iterator to the fluid particles to add to vectorDist
	auto fluid_it = DrawParticles::DrawBox(vectorDist,sz,domain,fluid_box);

	// here we fill some of the constants needed by the simulation
	max_fluid_height = fluid_it.getBoxMargins().getHigh(2);
	h_swl = fluid_it.getBoxMargins().getHigh(2) - fluid_it.getBoxMargins().getLow(2);
	B = (coeff_sound)*(coeff_sound)*gravity*h_swl*RhoZero / gamma_;
	cbar = coeff_sound * sqrt(gravity * h_swl);

	// for each particle inside the fluid box ...
	while (fluid_it.isNext())
	{
		// ... add a particle ...
		vectorDist.add();

		// ... and set it position ...
		vectorDist.getLastPos()[0] = fluid_it.get().get(0);
		vectorDist.getLastPos()[1] = fluid_it.get().get(1);
		vectorDist.getLastPos()[2] = fluid_it.get().get(2);

		// and its type.
		vectorDist.template getLastProp<TYPE>() = FLUID;

		// We also initialize the density of the particle and the hydro-static pressure given by
		//
		// RhoZero*g*h = P
		//
		// rho_p = (P/B + 1)^(1/Gamma) * RhoZero
		//

		vectorDist.template getLastProp<PRESSURE>() = RhoZero * gravity *  (max_fluid_height - fluid_it.get().get(2));

		vectorDist.template getLastProp<RHO>() = pow(vectorDist.template getLastProp<PRESSURE>() / B + 1, 1.0/gamma_) * RhoZero;
		vectorDist.template getLastProp<RHO_PREV>() = vectorDist.template getLastProp<RHO>();
		vectorDist.template getLastProp<VELOCITY>()[0] = 0.0;
		vectorDist.template getLastProp<VELOCITY>()[1] = 0.0;
		vectorDist.template getLastProp<VELOCITY>()[2] = 0.0;

		vectorDist.template getLastProp<VELOCITY_PREV>()[0] = 0.0;
		vectorDist.template getLastProp<VELOCITY_PREV>()[1] = 0.0;
		vectorDist.template getLastProp<VELOCITY_PREV>()[2] = 0.0;

		// next fluid particle
		++fluid_it;
	}

	// Recipient
	Box<3,real_number> recipient1({0.0f,0.0f,0.0f},{1.6f+dp/2.0f,0.67f+dp/2.0f,0.4f+dp/2.0f});
	Box<3,real_number> recipient2({dp,dp,dp},{1.6f-dp/2.0f,0.67f-dp/2.0f,0.4f+dp/2.0f});

	Box<3,real_number> obstacle1({0.9f,0.24f-dp/2.0f,0.0f},{1.02f+dp/2.0f,0.36f,0.45f+dp/2.0f});
	Box<3,real_number> obstacle2({0.9f+dp,0.24f+dp/2.0f,0.0f},{1.02f-dp/2.0f,0.36f-dp,0.45f-dp/2.0f});
	Box<3,real_number> obstacle3({0.9f+dp,0.24f,0.0f},{1.02f,0.36f,0.45f});

	openfpm::vector<Box<3,real_number>> holes;
	holes.add(recipient2);
	holes.add(obstacle1);
	auto bound_box = DrawParticles::DrawSkin(vectorDist,sz,domain,holes,recipient1);

	while (bound_box.isNext())
	{
		vectorDist.add();

		vectorDist.getLastPos()[0] = bound_box.get().get(0);
		vectorDist.getLastPos()[1] = bound_box.get().get(1);
		vectorDist.getLastPos()[2] = bound_box.get().get(2);

		vectorDist.template getLastProp<TYPE>() = BOUNDARY;
		vectorDist.template getLastProp<RHO>() = RhoZero;
		vectorDist.template getLastProp<RHO_PREV>() = RhoZero;
		vectorDist.template getLastProp<VELOCITY>()[0] = 0.0;
		vectorDist.template getLastProp<VELOCITY>()[1] = 0.0;
		vectorDist.template getLastProp<VELOCITY>()[2] = 0.0;

		vectorDist.template getLastProp<VELOCITY_PREV>()[0] = 0.0;
		vectorDist.template getLastProp<VELOCITY_PREV>()[1] = 0.0;
		vectorDist.template getLastProp<VELOCITY_PREV>()[2] = 0.0;

		++bound_box;
	}

	auto obstacle_box = DrawParticles::DrawSkin(vectorDist,sz,domain,obstacle2,obstacle1);

	while (obstacle_box.isNext())
	{
		vectorDist.add();

		vectorDist.getLastPos()[0] = obstacle_box.get().get(0);
		vectorDist.getLastPos()[1] = obstacle_box.get().get(1);
		vectorDist.getLastPos()[2] = obstacle_box.get().get(2);

		vectorDist.template getLastProp<TYPE>() = BOUNDARY;
		vectorDist.template getLastProp<RHO>() = RhoZero;
		vectorDist.template getLastProp<RHO_PREV>() = RhoZero;
		vectorDist.template getLastProp<VELOCITY>()[0] = 0.0;
		vectorDist.template getLastProp<VELOCITY>()[1] = 0.0;
		vectorDist.template getLastProp<VELOCITY>()[2] = 0.0;

		vectorDist.template getLastProp<VELOCITY_PREV>()[0] = 0.0;
		vectorDist.template getLastProp<VELOCITY_PREV>()[1] = 0.0;
		vectorDist.template getLastProp<VELOCITY_PREV>()[2] = 0.0;

		++obstacle_box;
	}

	vectorDist.map();

	// Now that we fill the vector with particles
	ModelCustom md;

	vectorDist.addComputationCosts(md);
	vectorDist.getDecomposition().decompose();
	vectorDist.map();

	///////////////////////////

	// Ok the initialization is done on CPU on GPU we are doing the main loop, so first we offload all properties on GPU

	vectorDist.hostToDevicePos();
	vectorDist.template hostToDeviceProp<TYPE,RHO,RHO_PREV,PRESSURE,VELOCITY,VELOCITY_PREV>();

	vectorDist.ghost_get<TYPE,RHO,PRESSURE,VELOCITY>(RUN_ON_DEVICE);

	auto cellList = vectorDist.getCellListGPU(2*H, CL_NON_SYMMETRIC, 2);

	timer tot_sim;
	tot_sim.start();

	size_t write = 0;
	size_t it = 0;
	size_t it_reb = 0;
	real_number t = 0.0;
	while (t <= t_end)
	{
		Vcluster<> & v_cl = create_vcluster();
		timer it_time;
		it_time.start();

		////// Do rebalancing every 200 timesteps
		it_reb++;
		if (it_reb == 300)
		{
			vectorDist.map(RUN_ON_DEVICE);

			// Rebalancer for now work on CPU , so move to CPU
			vectorDist.deviceToHostPos();
			vectorDist.template deviceToHostProp<TYPE>();

			it_reb = 0;
			ModelCustom md;
			vectorDist.addComputationCosts(md);
			vectorDist.getDecomposition().decompose();

			if (v_cl.getProcessUnitID() == 0)
			{std::cout << "REBALANCED " << it_reb << std::endl;}
		}

		vectorDist.map(RUN_ON_DEVICE);

		// Calculate pressure from the density
		EqState(vectorDist);

		real_number max_visc = 0.0;

		vectorDist.ghost_get<TYPE,RHO,PRESSURE,VELOCITY>(RUN_ON_DEVICE);

		// Calc forces
		calc_forces(vectorDist,cellList,max_visc,cnt);

		// Get the maximum viscosity term across processors
		v_cl.max(max_visc);
		v_cl.execute();

		// Calculate delta t integration
		real_number dt = calc_deltaT(vectorDist,max_visc);

		// VerletStep or euler step
		it++;
		if (it < 40)
			verlet_int(vectorDist,dt);
		else
		{
			euler_int(vectorDist,dt);
			it = 0;
		}

		t += dt;

		if (write < t*100)
		{
			// Sensor pressure require update ghost, so we ensure that particles are distributed correctly
			// and ghost are updated
			vectorDist.map(RUN_ON_DEVICE);
			vectorDist.ghost_get<TYPE,RHO,PRESSURE,VELOCITY>(RUN_ON_DEVICE);

			// calculate the pressure at the sensor points
			//sensor_pressure(vectorDist,cellList,press_t,probes);

			std::cout << "OUTPUT " << dt << std::endl;

			// When we write we have move all the particles information back to CPU

			vectorDist.deviceToHostPos();
			vectorDist.deviceToHostProp<TYPE,RHO,RHO_PREV,PRESSURE,DRHO,FORCE,VELOCITY,VELOCITY_PREV,RED,RED2>();

			vectorDist.write_frame("Geometry",write);
			write++;

			if (v_cl.getProcessUnitID() == 0)
			{std::cout << "TIME: " << t << "  write " << it_time.getwct() << "   " << it_reb << "   " << cnt << " Max visc: " << max_visc << "   " << vectorDist.size_local()  << std::endl;}
		}
		else
		{
			if (v_cl.getProcessUnitID() == 0)
			{std::cout << "TIME: " << t << "  " << it_time.getwct() << "   " << it_reb << "   " << cnt  << " Max visc: " << max_visc << "   " << vectorDist.size_local() << std::endl;}
		}
	}

	tot_sim.stop();
	std::cout << "Time to complete: " << tot_sim.getwct() << " seconds" << std::endl;

	openfpm_finalize();
}

#else

int main(int argc, char* argv[])
{
        return 0;
}

#endif
